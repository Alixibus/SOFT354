#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

__global__ void add(int* a, int* b, int* c)
{
	*c = *a + *b;
}

int main(void)
{
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = 2;
	b = 7;

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add << <1, 1 >> > (d_a, d_b, d_c);

	hipError_t error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("Error: %s\n", hipGetErrorString(error));
	}

	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	printf("The sum of %d and %d is equal %d\n", a, b, c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	hipDeviceReset();
	return 0;



}